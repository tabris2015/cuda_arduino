
#include "hip/hip_runtime.h"
#include ""

#include <Windows.h>
#include <stdio.h>

int main()
{
	HANDLE hSerial;
	DCB dcbSerialParams = {0};
	COMMTIMEOUTS timeouts = {0};


	fprintf(stderr, "abriendo puerto serial...");
	hSerial = CreateFile(
						"\\\\.\\COM21", GENERIC_READ|GENERIC_WRITE, 0, NULL,
						OPEN_EXISTING, FILE_ATTRIBUTE_NORMAL, NULL);

	if(hSerial == INVALID_HANDLE_VALUE)
	{
		fprintf(stderr, "Error\n");
		return 1;
	}
	else fprintf(stderr, "OK\n");

	//set device params ( 9600 baud
	dcbSerialParams.DCBlength = sizeof(dcbSerialParams);
    if (GetCommState(hSerial, &dcbSerialParams) == 0)
    {
        fprintf(stderr, "Error getting device state\n");
        CloseHandle(hSerial);
        return 1;
    }
     
	dcbSerialParams.BaudRate = CBR_9600;
    dcbSerialParams.ByteSize = 8;
    dcbSerialParams.StopBits = ONESTOPBIT;
    dcbSerialParams.Parity = NOPARITY;
    if(SetCommState(hSerial, &dcbSerialParams) == 0)
    {
        fprintf(stderr, "Error setting device parameters\n");
        CloseHandle(hSerial);
        return 1;
    }
 
    // Set COM port timeout settings
    timeouts.ReadIntervalTimeout = 50;
    timeouts.ReadTotalTimeoutConstant = 50;
    timeouts.ReadTotalTimeoutMultiplier = 10;
    timeouts.WriteTotalTimeoutConstant = 50;
    timeouts.WriteTotalTimeoutMultiplier = 10;
    if(SetCommTimeouts(hSerial, &timeouts) == 0)
    {
        fprintf(stderr, "Error setting timeouts\n");
        CloseHandle(hSerial);
        return 1;
    }
 
    // Send specified text (remaining command line arguments)
    DWORD tamano = 10;
	char bytes_recibidos[10];

	BYTE dato[10];
	DWORD temp;

    
	for(int i = 0; i<200000; i++)
	{
		if(!ReadFile(hSerial, &dato, tamano, &temp, NULL))
		{
			fprintf(stderr, "Error\n");
			CloseHandle(hSerial);
			return 1;
		}
   
		//fprintf(stderr, "%d recibidos: \n", tamano);
		for( int j = 0; j < 5; j++) printf("%c", dato[j]);
		printf("\n");
	}
    // Close serial port
    fprintf(stderr, "Closing serial port...");
    if (CloseHandle(hSerial) == 0)
    {
        fprintf(stderr, "Error\n");
        return 1;
    }
    fprintf(stderr, "OK\n");
 
    // exit normally
	getchar();
    return 0;
}